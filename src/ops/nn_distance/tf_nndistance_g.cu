
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"
#include "float.h"
#define INNER_BATCH_SIZE 4
#define BATCH 512

__global__ void NmDistanceKernel(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i){
	__shared__ float buf[BATCH*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int k2=0;k2<m;k2+=BATCH){
			int end_k=min(m,k2+BATCH)-k2;
			for (int j=threadIdx.x;j<end_k*3;j+=blockDim.x){
				buf[j]=xyz2[(i*m+k2)*3+j];
			}
			__syncthreads();
			for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
				float x1=xyz[(i*n+j)*3+0];
				float y1=xyz[(i*n+j)*3+1];
				float z1=xyz[(i*n+j)*3+2];
				int best_i=0;
				float best=FLT_MAX;
				int end_ka=end_k-(end_k & (INNER_BATCH_SIZE - 1));
                for (int k=0;k<end_ka;k+=INNER_BATCH_SIZE){
                    #pragma unroll
                    for (int u=0; u < INNER_BATCH_SIZE; u++) {
                        float x2=buf[k*3+u*3]-x1;
                        float y2=buf[k*3+u*3+1]-y1;
                        float z2=buf[k*3+u*3+2]-z1;
                        float d=x2*x2+y2*y2+z2*z2;
                        if (d<best){
                            best=d;
                            best_i=k+k2+u;
                        }
                    }
                }
				for (int k=end_ka;k<end_k;k++){
					float x2=buf[k*3+0]-x1;
					float y2=buf[k*3+1]-y1;
					float z2=buf[k*3+2]-z1;
					float d=x2*x2+y2*y2+z2*z2;
					if (k==0 || d<best){
						best=d;
						best_i=k+k2;
					}
				}
				if (k2==0 || result[(i*n+j)]>best){
					result[(i*n+j)]=best;
					result_i[(i*n+j)]=best_i;
				}
			}
			__syncthreads();
		}
	}
}
void NmDistanceKernelLauncher(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i,float * result2,int * result2_i){
	NmDistanceKernel<<<dim3(4,128,1),512>>>(b,n,xyz,m,xyz2,result,result_i);
	NmDistanceKernel<<<dim3(4,128,1),512>>>(b,m,xyz2,n,xyz,result2,result2_i);
}
__global__ void NmDistanceGradKernel(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,float * grad_xyz1,float * grad_xyz2){
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
			float x1=xyz1[(i*n+j)*3+0];
			float y1=xyz1[(i*n+j)*3+1];
			float z1=xyz1[(i*n+j)*3+2];
			int j2=idx1[i*n+j];
			float x2=xyz2[(i*m+j2)*3+0];
			float y2=xyz2[(i*m+j2)*3+1];
			float z2=xyz2[(i*m+j2)*3+2];
			float g=grad_dist1[i*n+j]*2;
			atomicAdd(&(grad_xyz1[(i*n+j)*3+0]),g*(x1-x2));
			atomicAdd(&(grad_xyz1[(i*n+j)*3+1]),g*(y1-y2));
			atomicAdd(&(grad_xyz1[(i*n+j)*3+2]),g*(z1-z2));
			atomicAdd(&(grad_xyz2[(i*m+j2)*3+0]),-(g*(x1-x2)));
			atomicAdd(&(grad_xyz2[(i*m+j2)*3+1]),-(g*(y1-y2)));
			atomicAdd(&(grad_xyz2[(i*m+j2)*3+2]),-(g*(z1-z2)));
		}
	}
}
void NmDistanceGradKernelLauncher(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,const float * grad_dist2,const int * idx2,float * grad_xyz1,float * grad_xyz2){
	cudaMemset(grad_xyz1,0,b*n*3*4);
	cudaMemset(grad_xyz2,0,b*m*3*4);
	NmDistanceGradKernel<<<dim3(1,64,1),512>>>(b,n,xyz1,m,xyz2,grad_dist1,idx1,grad_xyz1,grad_xyz2);
	NmDistanceGradKernel<<<dim3(1,64,1),512>>>(b,m,xyz2,n,xyz1,grad_dist2,idx2,grad_xyz2,grad_xyz1);
}

#endif
